
#include <hip/hip_runtime.h>
#include <iostream>

// MISC. UTILITY FUNCTIONS

// Error handler for CUDA GPU calls. 
//   Returns error number, filename and line number containing the error to the terminal.
//   Please refer to CUDA_Toolkit_Reference_Manual.pdf, section 4.23.3.3 enum cudaError
//   for error discription. Error enumeration starts from 0.
void checkForCudaErrors(const char* checkpoint_description)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "\nCuda error detected, checkpoint: " << checkpoint_description
            << "\nError string: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkForCudaErrors(const char* checkpoint_description, const unsigned int iteration)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "\nCuda error detected, checkpoint: " << checkpoint_description
            << "\nduring iteration " << iteration
            << "\nError string: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// vim: tabstop=8 expandtab shiftwidth=4 softtabstop=4
