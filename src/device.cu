#include "hip/hip_runtime.h"
// device.cu -- GPU specific operations utilizing the CUDA API.
#include <iostream>
#include <string>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cutil_math.h>

#include "vector_arithmetic.h"	// for arbitrary prec. vectors
//#include <vector_functions.h>	// for single prec. vectors
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

#include "sphere.h"
#include "datatypes.h"
#include "utility.cuh"
#include "constants.cuh"
#include "debug.h"

#include "sorting.cuh"	
#include "contactmodels.cuh"
#include "cohesion.cuh"
#include "contactsearch.cuh"
#include "integration.cuh"
#include "raytracer.cuh"

//#include "cuPrintf.cu"

// Wrapper function for initializing the CUDA components.
// Called from main.cpp
//extern "C"
__host__ void DEM::initializeGPU(void)
{
  using std::cout; // stdout

  // Specify target device
  int cudadevice = 0;

  // Variables containing device properties
  hipDeviceProp_t prop;
  int devicecount;
  int cudaDriverVersion;
  int cudaRuntimeVersion;


  // Register number of devices
  hipGetDeviceCount(&devicecount);

  if(devicecount == 0) {
    std::cerr << "\nERROR: No CUDA-enabled devices availible. Bye."
      << std::endl;
    exit(EXIT_FAILURE);
  } else if (devicecount == 1) {
    if (verbose == 1)
      cout << "  System contains 1 CUDA compatible device.\n";
  } else {
    if (verbose == 1)
      cout << "  System contains " << devicecount << " CUDA compatible devices.\n";
  }

  hipGetDeviceProperties(&prop, cudadevice);
  hipDriverGetVersion(&cudaDriverVersion);
  hipRuntimeGetVersion(&cudaRuntimeVersion);

  if (verbose == 1) {
    cout << "  Using CUDA device ID: " << cudadevice << "\n";
    cout << "  - Name: " <<  prop.name << ", compute capability: " 
      << prop.major << "." << prop.minor << ".\n";
    cout << "  - CUDA Driver version: " << cudaDriverVersion/1000 
      << "." <<  cudaDriverVersion%100 
      << ", runtime version " << cudaRuntimeVersion/1000 << "." 
      << cudaRuntimeVersion%100 << std::endl;
  }

  // Comment following line when using a system only containing exclusive mode GPUs
  hipChooseDevice(&cudadevice, &prop); 

  checkForCudaErrors("After initializing CUDA device");
}

// Start timer for kernel profiling
__host__ void startTimer(hipEvent_t* kernel_tic)
{
  hipEventRecord(*kernel_tic);
}

// Stop timer for kernel profiling and time to function sum
__host__ void stopTimer(hipEvent_t *kernel_tic,
    			hipEvent_t *kernel_toc,
			float *kernel_elapsed,
			double* sum)
{
    hipEventRecord(*kernel_toc, 0);
    hipEventSynchronize(*kernel_toc);
    hipEventElapsedTime(kernel_elapsed, *kernel_tic, *kernel_toc);
    *sum += *kernel_elapsed;
}

// Check values of parameters in constant memory
__global__ void checkConstantValues(int* dev_equal,
    				    Grid* dev_grid,
				    Params* dev_params)
{

  // Values ok (0)
  *dev_equal = 0;

  // Compare values between global- and constant
  // memory structures
  if (dev_grid->origo[0] != devC_grid.origo[0] ||
      dev_grid->origo[1] != devC_grid.origo[1] ||
      dev_grid->origo[2] != devC_grid.origo[2] ||
      dev_grid->L[0] != devC_grid.L[0] ||
      dev_grid->L[1] != devC_grid.L[1] ||
      dev_grid->L[2] != devC_grid.L[2] ||
      dev_grid->num[0] != devC_grid.num[0] ||
      dev_grid->num[1] != devC_grid.num[1] ||
      dev_grid->num[2] != devC_grid.num[2] ||
      dev_grid->periodic != devC_grid.periodic)
    *dev_equal = 1; // Not ok

  
  else if (dev_params->g[0] != devC_params.g[0] ||
      dev_params->g[1] != devC_params.g[1] ||
      dev_params->g[2] != devC_params.g[2] ||
      dev_params->k_n != devC_params.k_n ||
      dev_params->k_t != devC_params.k_t ||
      dev_params->k_r != devC_params.k_r ||
      dev_params->gamma_n != devC_params.gamma_n ||
      dev_params->gamma_t != devC_params.gamma_t ||
      dev_params->gamma_r != devC_params.gamma_r ||
      dev_params->mu_s != devC_params.mu_s ||
      dev_params->mu_d != devC_params.mu_d ||
      dev_params->mu_r != devC_params.mu_r ||
      dev_params->rho != devC_params.rho ||
      dev_params->contactmodel != devC_params.contactmodel ||
      dev_params->kappa != devC_params.kappa ||
      dev_params->db != devC_params.db ||
      dev_params->V_b != devC_params.V_b)
    *dev_equal = 2; // Not ok

}


// Copy the constant data components to device memory,
// and check whether the values correspond to the 
// values in constant memory.
__host__ void DEM::checkConstantMemory()
{

  //cudaPrintfInit();

  // Allocate space in global device memory
  Grid* dev_grid;
  Params* dev_params;
  hipMalloc((void**)&dev_grid, sizeof(Grid));
  hipMalloc((void**)&dev_params, sizeof(Params));

  // Copy structure data from host to global device memory
  hipMemcpy(dev_grid, &grid, sizeof(Grid), hipMemcpyHostToDevice);
  hipMemcpy(dev_params, &params, sizeof(Params), hipMemcpyHostToDevice);

  // Compare values between global and constant memory
  // structures on the device.
  int* equal = new int;	// The values are equal = 0, if not = 1
  *equal = 0;
  int* dev_equal;
  hipMalloc((void**)&dev_equal, sizeof(int));
  checkConstantValues<<<1,1>>>(dev_equal, dev_grid, dev_params);
  checkForCudaErrors("After constant memory check");

  // Copy result to host
  hipMemcpy(equal, dev_equal, sizeof(int), hipMemcpyDeviceToHost);

  // Free global device memory
  hipFree(dev_grid);
  hipFree(dev_params);
  hipFree(dev_equal);

  //cudaPrintfDisplay(stdout, true);

  // Are the values equal?
  if (*equal != 0) {
    std::cerr << "Error! The values in constant memory do not "
              << "seem to be correct (" << *equal << ").\n";
    exit(1);
  } else {
    std::cout << "  Constant values ok (" << *equal << ").\n";
  }
}

// Copy selected constant components to constant device memory.
__host__ void DEM::transferToConstantDeviceMemory()
{
  using std::cout;

  if (verbose == 1)
    cout << "\n  Transfering data to constant device memory:     ";

  hipMemcpyToSymbol(HIP_SYMBOL("devC_nd"), &nd, sizeof(nd));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_np"), &np, sizeof(np));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_nw"), &walls.nw, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_nc"), &NC, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_dt"), &time.dt, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(devC_grid), &grid, sizeof(Grid));
  hipMemcpyToSymbol(HIP_SYMBOL(devC_params), &params, sizeof(Params));
  
  checkForCudaErrors("After transferring to device constant memory");
  
  if (verbose == 1)
    cout << "Done\n";

  checkConstantMemory();
}


// Allocate device memory for particle variables,
// tied to previously declared pointers in structures
__host__ void DEM::allocateGlobalDeviceMemory(void)
{
  // Particle memory size
  unsigned int memSizeF  = sizeof(Float) * np;
  unsigned int memSizeF4 = sizeof(Float4) * np;

  if (verbose == 1)
    std::cout << "  Allocating global device memory:                ";

  k.acc = new Float4[np];
  k.angacc = new Float4[np];

  // Kinematics arrays
  hipMalloc((void**)&dev_x, memSizeF4);
  hipMalloc((void**)&dev_xysum, memSizeF4);
  hipMalloc((void**)&dev_vel, memSizeF4);
  hipMalloc((void**)&dev_acc, memSizeF4);
  hipMalloc((void**)&dev_force, memSizeF4);
  hipMalloc((void**)&dev_angpos, memSizeF4);
  hipMalloc((void**)&dev_angvel, memSizeF4);
  hipMalloc((void**)&dev_angacc, memSizeF4);
  hipMalloc((void**)&dev_torque, memSizeF4);

  // Particle contact bookkeeping arrays
  hipMalloc((void**)&dev_contacts, sizeof(unsigned int)*np*NC); // Max NC contacts per particle
  hipMalloc((void**)&dev_distmod, memSizeF4*NC);
  hipMalloc((void**)&dev_delta_t, memSizeF4*NC);

  // Sorted arrays
  hipMalloc((void**)&dev_x_sorted, memSizeF4);
  hipMalloc((void**)&dev_vel_sorted, memSizeF4);
  hipMalloc((void**)&dev_angvel_sorted, memSizeF4);

  // Energy arrays
  hipMalloc((void**)&dev_es_dot, memSizeF);
  hipMalloc((void**)&dev_ev_dot, memSizeF);
  hipMalloc((void**)&dev_es, memSizeF);
  hipMalloc((void**)&dev_ev, memSizeF);
  hipMalloc((void**)&dev_p, memSizeF);

  // Cell-related arrays
  hipMalloc((void**)&dev_gridParticleCellID, sizeof(unsigned int)*np);
  hipMalloc((void**)&dev_gridParticleIndex, sizeof(unsigned int)*np);
  hipMalloc((void**)&dev_cellStart, sizeof(unsigned int)*grid.num[0]*grid.num[1]*grid.num[2]);
  hipMalloc((void**)&dev_cellEnd, sizeof(unsigned int)*grid.num[0]*grid.num[1]*grid.num[2]);

    // Host contact bookkeeping arrays
  k.contacts = new unsigned int[np*NC];
  // Initialize contacts lists to np
  for (unsigned int i=0; i<(np*NC); ++i)
    k.contacts[i] = np;
  k.distmod = new Float4[np*NC];
  k.delta_t = new Float4[np*NC];

  // Wall arrays
  hipMalloc((void**)&dev_walls_wmode, sizeof(int)*walls.nw);
  hipMalloc((void**)&dev_walls_nx, sizeof(Float4)*walls.nw);
  hipMalloc((void**)&dev_walls_mvfd, sizeof(Float4)*walls.nw);
  hipMalloc((void**)&dev_walls_force_pp, sizeof(Float)*walls.nw*np);
  // dev_walls_force_partial allocated later

  checkForCudaErrors("End of allocateGlobalDeviceMemory");
  if (verbose == 1)
    std::cout << "Done\n";
}

__host__ void DEM::freeGlobalDeviceMemory()
{
  if (verbose == 1)
    printf("\nLiberating device memory:                        ");
  // Particle arrays
  hipFree(dev_x);
  hipFree(dev_xysum);
  hipFree(dev_vel);
  hipFree(dev_acc);
  hipFree(dev_force);
  hipFree(dev_angpos);
  hipFree(dev_angvel);
  hipFree(dev_angacc);
  hipFree(dev_torque);

  hipFree(dev_contacts);
  hipFree(dev_distmod);
  hipFree(dev_delta_t);

  hipFree(dev_es_dot);
  hipFree(dev_es);
  hipFree(dev_ev_dot);
  hipFree(dev_ev);
  hipFree(dev_p);

  hipFree(dev_x_sorted);
  hipFree(dev_vel_sorted);
  hipFree(dev_angvel_sorted);

  // Cell-related arrays
  hipFree(dev_gridParticleIndex);
  hipFree(dev_cellStart);
  hipFree(dev_cellEnd);

  // Wall arrays
  hipFree(dev_walls_nx);
  hipFree(dev_walls_mvfd);
  hipFree(dev_walls_force_partial);
  hipFree(dev_walls_force_pp);

  if (verbose == 1)
    printf("Done\n");
}


__host__ void DEM::transferToGlobalDeviceMemory()
{
  if (verbose == 1)
    std::cout << "  Transfering data to the device:                 ";

  // Commonly-used memory sizes
  unsigned int memSizeF  = sizeof(Float) * np;
  unsigned int memSizeF4 = sizeof(Float4) * np;

  // Copy static-size structure data from host to global device memory
  //hipMemcpy(dev_time, &time, sizeof(Time), hipMemcpyHostToDevice);

  // Kinematic particle values
  hipMemcpy( dev_x,	       k.x,	   
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_xysum,    k.xysum,
      sizeof(Float2)*np, hipMemcpyHostToDevice);
  hipMemcpy( dev_vel,      k.vel,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_acc,      k.acc, 
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_force,    k.force,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_angpos,   k.angpos,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_angvel,   k.angvel,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_angacc,   k.angacc,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_torque,   k.torque,
      memSizeF4, hipMemcpyHostToDevice);
  hipMemcpy( dev_contacts, k.contacts,
      sizeof(unsigned int)*np*NC, hipMemcpyHostToDevice);
  hipMemcpy( dev_distmod, k.distmod,
      memSizeF4*NC, hipMemcpyHostToDevice);
  hipMemcpy( dev_delta_t, k.delta_t,
      memSizeF4*NC, hipMemcpyHostToDevice);

  // Individual particle energy values
  hipMemcpy( dev_es_dot, e.es_dot,
      memSizeF, hipMemcpyHostToDevice);
  hipMemcpy( dev_es,     e.es,
      memSizeF, hipMemcpyHostToDevice);
  hipMemcpy( dev_ev_dot, e.ev_dot,
      memSizeF, hipMemcpyHostToDevice);
  hipMemcpy( dev_ev,     e.ev,
      memSizeF, hipMemcpyHostToDevice);
  hipMemcpy( dev_p, e.p,
      memSizeF, hipMemcpyHostToDevice);

  // Wall parameters
  hipMemcpy( dev_walls_wmode, walls.wmode,
      sizeof(int)*walls.nw, hipMemcpyHostToDevice);
  hipMemcpy( dev_walls_nx,    walls.nx,
      sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);
  hipMemcpy( dev_walls_mvfd,  walls.mvfd,
      sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);

  checkForCudaErrors("End of transferToGlobalDeviceMemory");
  if (verbose == 1)
    std::cout << "Done\n";
}

__host__ void DEM::transferFromGlobalDeviceMemory()
{
  //std::cout << "  Transfering data from the device:               ";

  // Commonly-used memory sizes
  unsigned int memSizeF  = sizeof(Float) * np;
  unsigned int memSizeF4 = sizeof(Float4) * np;

  // Copy static-size structure data from host to global device memory
  //hipMemcpy(&time, dev_time, sizeof(Time), hipMemcpyDeviceToHost);

  // Kinematic particle values
  hipMemcpy( k.x, dev_x,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.xysum, dev_xysum,
      sizeof(Float2)*np, hipMemcpyDeviceToHost);
  hipMemcpy( k.vel, dev_vel,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.acc, dev_acc,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.force, dev_force,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.angpos, dev_angpos,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.angvel, dev_angvel,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.angacc, dev_angacc,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.torque, dev_torque,
      memSizeF4, hipMemcpyDeviceToHost);
  hipMemcpy( k.contacts, dev_contacts,
      sizeof(unsigned int)*np*NC, hipMemcpyDeviceToHost);
  hipMemcpy( k.distmod, dev_distmod,
      memSizeF4*NC, hipMemcpyDeviceToHost);
  hipMemcpy( k.delta_t, dev_delta_t,
      memSizeF4*NC, hipMemcpyDeviceToHost);

  // Individual particle energy values
  hipMemcpy( e.es_dot, dev_es_dot,
      memSizeF, hipMemcpyDeviceToHost);
  hipMemcpy( e.es, dev_es,
      memSizeF, hipMemcpyDeviceToHost);
  hipMemcpy( e.ev_dot, dev_ev_dot,
      memSizeF, hipMemcpyDeviceToHost);
  hipMemcpy( e.ev, dev_ev,
      memSizeF, hipMemcpyDeviceToHost);
  hipMemcpy( e.p, dev_p,
      memSizeF, hipMemcpyDeviceToHost);

  // Wall parameters
  hipMemcpy( walls.wmode, dev_walls_wmode,
      sizeof(int)*walls.nw, hipMemcpyDeviceToHost);
  hipMemcpy( walls.nx, dev_walls_nx,
      sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);
  hipMemcpy( walls.mvfd, dev_walls_mvfd,
      sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);

  checkForCudaErrors("End of transferFromGlobalDeviceMemory");
}


// Iterate through time by explicit time integration
__host__ void DEM::startTime()
{
  using std::cout; // Namespace directive
  std::string outfile;
  char file[200];
  FILE *fp;

  // Synchronization point
  hipDeviceSynchronize();
  checkForCudaErrors("Start of startTime()");

  // Model world variables
  float tic, toc, filetimeclock, time_spent, dev_time_spent;

  // Start CPU clock
  tic = clock();

  // GPU workload configuration
  unsigned int threadsPerBlock = 256; 
  // Create enough blocks to accomodate the particles
  unsigned int blocksPerGrid = iDivUp(np, threadsPerBlock); 
  dim3 dimGrid(blocksPerGrid, 1, 1); // Blocks arranged in 1D grid
  dim3 dimBlock(threadsPerBlock, 1, 1); // Threads arranged in 1D block
  // Shared memory per block
  unsigned int smemSize = sizeof(unsigned int)*(threadsPerBlock+1);

  // Pre-sum of force per wall
  hipMalloc((void**)&dev_walls_force_partial, sizeof(Float)*dimGrid.x);

  // Report to stdout
  if (verbose == 1) {
    cout << "\n  Device memory allocation and transfer complete.\n"
      << "  - Blocks per grid: "
      << dimGrid.x << "*" << dimGrid.y << "*" << dimGrid.z << "\n"
      << "  - Threads per block: "
      << dimBlock.x << "*" << dimBlock.y << "*" << dimBlock.z << "\n"
      << "  - Shared memory required per block: " << smemSize << " bytes\n";
  }

  // Initialize counter variable values
  filetimeclock = 0.0;
  long iter = 0;

  // Create first status.dat
  //sprintf(file,"output/%s.status.dat", sid);
  outfile = "output/" + sid + ".status.dat";
  fp = fopen(outfile.c_str(), "w");
  fprintf(fp,"%2.4e %2.4e %d\n", 
      	  time.current, 
	  100.0*time.current/time.total, 
	  time.step_count);
  fclose(fp);

  // Write first output data file: output0.bin, thus testing writing of bin files
  outfile = "output/" + sid + ".output0.bin";
  //sprintf(file,"output/%s.output0.bin", sid);
  writebin(outfile.c_str());

  if (verbose == 1) {
    cout << "\n  Entering the main calculation time loop...\n\n"
      << "  IMPORTANT: Do not close this terminal, doing so will \n"
      << "             terminate this SPHERE process. Follow the \n"
      << "             progress by executing:\n"
      << "                $ ./sphere_status " << sid << "\n\n";
  }


  // Start GPU clock
  hipEvent_t dev_tic, dev_toc;
  hipEventCreate(&dev_tic);
  hipEventCreate(&dev_toc);
  hipEventRecord(dev_tic, 0);

  // If profiling is enabled, initialize timers for each kernel
  hipEvent_t kernel_tic, kernel_toc;
  float kernel_elapsed;
  double t_calcParticleCellID = 0.0;
  double t_thrustsort = 0.0;
  double t_reorderArrays = 0.0;
  double t_topology = 0.0;
  double t_interact = 0.0;
  double t_integrate = 0.0;
  double t_summation = 0.0;
  double t_integrateWalls = 0.0;

  if (PROFILING == 1) {
    hipEventCreate(&kernel_tic);
    hipEventCreate(&kernel_toc);
  }

  cout << "  Current simulation time: " << time.current << " s.";


  // MAIN CALCULATION TIME LOOP
  while (time.current <= time.total) {

    // Increment iteration counter
    ++iter;

    // Print current step number to terminal
    //printf("Step: %d\n", time.step_count);


    // Routine check for errors
    checkForCudaErrors("Start of main while loop");


    // For each particle: 
    // Compute hash key (cell index) from position 
    // in the fine, uniform and homogenous grid.
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    calcParticleCellID<<<dimGrid, dimBlock>>>(dev_gridParticleCellID, 
					      dev_gridParticleIndex, 
					      dev_x);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_calcParticleCellID);
    checkForCudaErrors("Post calcParticleCellID");


    // Sort particle (key, particle ID) pairs by hash key with Thrust radix sort
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    thrust::sort_by_key(thrust::device_ptr<uint>(dev_gridParticleCellID),
			thrust::device_ptr<uint>(dev_gridParticleCellID + np),
			thrust::device_ptr<uint>(dev_gridParticleIndex));
    hipDeviceSynchronize(); // Needed? Does thrust synchronize threads implicitly?
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_thrustsort);
    checkForCudaErrors("Post thrust::sort_by_key");


    // Zero cell array values by setting cellStart to its highest possible value,
    // specified with pointer value 0xffffffff, which for a 32 bit unsigned int
    // is 4294967295.
    hipMemset(dev_cellStart, 0xffffffff, 
	       grid.num[0]*grid.num[1]*grid.num[2]*sizeof(unsigned int));
    hipDeviceSynchronize();
    checkForCudaErrors("Post hipMemset");

    // Use sorted order to reorder particle arrays (position, velocities, radii) to ensure
    // coherent memory access. Save ordered configurations in new arrays (*_sorted).
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    reorderArrays<<<dimGrid, dimBlock, smemSize>>>(dev_cellStart, 
						   dev_cellEnd,
						   dev_gridParticleCellID, 
						   dev_gridParticleIndex,
						   dev_x, dev_vel, 
						   dev_angvel,
						   dev_x_sorted, 
						   dev_vel_sorted, 
						   dev_angvel_sorted);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_reorderArrays);
    checkForCudaErrors("Post reorderArrays", iter);

    // The contact search in topology() is only necessary for determining
    // the accumulated shear distance needed in the linear elastic
    // and nonlinear contact force model
    if (params.contactmodel == 2 || params.contactmodel == 3) {
      // For each particle: Search contacts in neighbor cells
      if (PROFILING == 1)
	startTimer(&kernel_tic);
      topology<<<dimGrid, dimBlock>>>(dev_cellStart, 
				      dev_cellEnd,
				      dev_gridParticleIndex,
				      dev_x_sorted, 
				      dev_contacts,
				      dev_distmod);


      // Synchronization point
      hipDeviceSynchronize();
      if (PROFILING == 1)
	stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_topology);
      checkForCudaErrors("Post topology: One or more particles moved outside the grid.\nThis could possibly be caused by a numerical instability.\nIs the computational time step too large?", iter);
    }


    // For each particle: Process collisions and compute resulting forces.
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    interact<<<dimGrid, dimBlock>>>(dev_gridParticleIndex,
				    dev_cellStart,
				    dev_cellEnd,
				    dev_x,
				    dev_x_sorted,
				    dev_vel_sorted,
				    dev_angvel_sorted,
				    dev_vel,
				    dev_angvel,
				    dev_force, 
				    dev_torque, 
				    dev_es_dot,
				    dev_ev_dot, 
				    dev_es,
				    dev_ev,
				    dev_p,
				    dev_walls_nx,
				    dev_walls_mvfd,
				    dev_walls_force_pp,
				    dev_contacts,
				    dev_distmod,
				    dev_delta_t);


    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_interact);
    checkForCudaErrors("Post interact - often caused if particles move outside the grid", iter);

    // Update particle kinematics
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    integrate<<<dimGrid, dimBlock>>>(dev_x_sorted, 
				     dev_vel_sorted, 
				     dev_angvel_sorted,
				     dev_x, 
				     dev_vel, 
				     dev_angvel,
				     dev_force,
				     dev_torque, 
				     dev_angpos,
				     dev_xysum,
				     dev_gridParticleIndex);
    hipDeviceSynchronize();
    checkForCudaErrors("Post integrate");
 

    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_integrate);

    // Summation of forces on wall
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    if (walls.nw > 0) {
      summation<<<dimGrid, dimBlock>>>(dev_walls_force_pp,
				       dev_walls_force_partial);
    }
    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_summation);
    checkForCudaErrors("Post wall force summation");

    // Update wall kinematics
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    if (walls.nw > 0) {
      integrateWalls<<< 1, walls.nw>>>(dev_walls_nx,
				       dev_walls_mvfd,
				       dev_walls_wmode,
				       dev_walls_force_partial,
				       blocksPerGrid);
    }

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_integrateWalls);
    checkForCudaErrors("Post integrateWalls");


    // Update timers and counters
    time.current  += time.dt;
    filetimeclock += time.dt;

    // Report time to console
    if (verbose == 1) {
      cout << "\r  Current simulation time: " 
	<< time.current << " s.        ";// << std::flush;
    }


    // Produce output binary if the time interval 
    // between output files has been reached
    if (filetimeclock > time.file_dt) {

      // Pause the CPU thread until all CUDA calls previously issued are completed
      hipDeviceSynchronize();
      checkForCudaErrors("Beginning of file output section");

      //// Copy device data to host memory
      transferFromGlobalDeviceMemory();

      // Pause the CPU thread until all CUDA calls previously issued are completed
      hipDeviceSynchronize();

      // Write binary output file
      time.step_count += 1;
      sprintf(file,"output/%s.output%d.bin", sid.c_str(), time.step_count);
      writebin(file);


      if (CONTACTINFO == 1) {
	// Write contact information to stdout
	cout << "\n\n---------------------------\n"
	     << "t = " << time.current << " s.\n"
	     << "---------------------------\n";

	for (int n = 0; n < np; ++n) {
	  cout << "\n## Particle " << n << " ##\n";

	  cout  << "- contacts:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << "[" << nc << "]=" << k.contacts[nc+NC*n] << '\n';

	  cout << "\n- delta_t:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << k.delta_t[nc+NC*n].x << '\t'
		 << k.delta_t[nc+NC*n].y << '\t'
		 << k.delta_t[nc+NC*n].z << '\t'
		 << k.delta_t[nc+NC*n].w << '\n';

	  cout << "\n- distmod:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << k.distmod[nc+NC*n].x << '\t'
		 << k.distmod[nc+NC*n].y << '\t'
		 << k.distmod[nc+NC*n].z << '\t'
		 << k.distmod[nc+NC*n].w << '\n';
	}
	cout << '\n';
      }

      // Update status.dat at the interval of filetime 
      outfile = "output/" + sid + ".status.dat";
      fp = fopen(outfile.c_str(), "w");
      fprintf(fp,"%2.4e %2.4e %d\n", 
	      time.current, 
	      100.0*time.current/time.total,
	      time.step_count);
      fclose(fp);

      filetimeclock = 0.0;
    }
  }

  // Stop clock and display calculation time spent
  toc = clock();
  hipEventRecord(dev_toc, 0);
  hipEventSynchronize(dev_toc);

  time_spent = (toc - tic)/(CLOCKS_PER_SEC);
  hipEventElapsedTime(&dev_time_spent, dev_tic, dev_toc);

  cout << "\nSimulation ended. Statistics:\n"
       << "  - Last output file number: " 
       << time.step_count << "\n"
       << "  - GPU time spent: "
       << dev_time_spent/1000.0f << " s\n"
       << "  - CPU time spent: "
       << time_spent << " s\n"
       << "  - Mean duration of iteration:\n"
       << "      " << dev_time_spent/((double)iter*1000.0f) << " s\n"; 

  hipEventDestroy(dev_tic);
  hipEventDestroy(dev_toc);

  hipEventDestroy(kernel_tic);
  hipEventDestroy(kernel_toc);

  // Report time spent on each kernel
  if (PROFILING == 1) {
    double t_sum = t_calcParticleCellID + t_thrustsort + t_reorderArrays
                 + t_topology + t_interact + t_summation + t_integrateWalls;
    cout << "\nKernel profiling statistics:\n"
         << "  - calcParticleCellID:\t" << t_calcParticleCellID/1000.0 << " s"
	 << "\t(" << 100.0*t_calcParticleCellID/t_sum << " %)\n"
         << "  - thrustsort:\t\t" << t_thrustsort/1000.0 << " s"
	 << "\t(" << 100.0*t_thrustsort/t_sum << " %)\n"
         << "  - reorderArrays:\t" << t_reorderArrays/1000.0 << " s"
	 << "\t(" << 100.0*t_reorderArrays/t_sum << " %)\n"
         << "  - topology:\t\t" << t_topology/1000.0 << " s"
	 << "\t(" << 100.0*t_topology/t_sum << " %)\n"
         << "  - interact:\t\t" << t_interact/1000.0 << " s"
	 << "\t(" << 100.0*t_interact/t_sum << " %)\n"
         << "  - integrate:\t\t" << t_integrate/1000.0 << " s"
	 << "\t(" << 100.0*t_integrate/t_sum << " %)\n"
         << "  - summation:\t\t" << t_summation/1000.0 << " s"
	 << "\t(" << 100.0*t_summation/t_sum << " %)\n"
         << "  - integrateWalls:\t" << t_integrateWalls/1000.0 << " s"
	 << "\t(" << 100.0*t_integrateWalls/t_sum << " %)\n";
  }


  // Free GPU device memory  
  freeGlobalDeviceMemory();

  // Contact info arrays
  delete[] k.contacts;
  delete[] k.distmod;
  delete[] k.delta_t;

} /* EOF */
