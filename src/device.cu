#include "hip/hip_runtime.h"
// device.cu -- GPU specific operations utilizing the CUDA API.
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "vector_arithmetic.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

#include "sphere.h"
#include "datatypes.h"
#include "utility.cuh"
#include "constants.cuh"

#include "sorting.cuh"	
#include "contactmodels.cuh"
#include "cohesion.cuh"
#include "contactsearch.cuh"
#include "integration.cuh"

//#include "cuPrintf.cu"

// Wrapper function for initializing the CUDA components.
// Called from main.cpp
//extern "C"
__host__ void DEM::initializeGPU(void)
{
  using std::cout; // stdout

  // Specify target device
  int cudadevice = 0;

  // Variables containing device properties
  hipDeviceProp_t prop;
  int devicecount;
  int cudaDriverVersion;
  int cudaRuntimeVersion;


  // Register number of devices
  hipGetDeviceCount(&devicecount);

  if(devicecount == 0) {
    std::cerr << "\nERROR: No CUDA-enabled devices availible. Bye.\n";
    exit(EXIT_FAILURE);
  } else if (devicecount == 1) {
    cout << "\nSystem contains 1 CUDA compatible device.\n";
  } else {
    cout << "\nSystem contains " << devicecount << " CUDA compatible devices.\n";
  }

  hipGetDeviceProperties(&prop, cudadevice);
  hipDriverGetVersion(&cudaDriverVersion);
  hipRuntimeGetVersion(&cudaRuntimeVersion);

  cout << "Using CUDA device ID: " << cudadevice << "\n";
  cout << "  - Name: " <<  prop.name << ", compute capability: " 
    << prop.major << "." << prop.minor << ".\n";
  cout << "  - CUDA Driver version: " << cudaDriverVersion/1000 
    << "." <<  cudaDriverVersion%100 
    << ", runtime version " << cudaRuntimeVersion/1000 << "." 
    << cudaRuntimeVersion%100 << "\n\n";

  // Comment following line when using a system only containing exclusive mode GPUs
  hipChooseDevice(&cudadevice, &prop); 

  checkForCudaErrors("After initializing CUDA device");
}

// Start timer for kernel profiling
__host__ void startTimer(hipEvent_t* kernel_tic)
{
  hipEventRecord(*kernel_tic);
}

// Stop timer for kernel profiling and time to function sum
__host__ void stopTimer(hipEvent_t *kernel_tic,
    			hipEvent_t *kernel_toc,
			float *kernel_elapsed,
			double* sum)
{
    hipEventRecord(*kernel_toc, 0);
    hipEventSynchronize(*kernel_toc);
    hipEventElapsedTime(kernel_elapsed, *kernel_tic, *kernel_toc);
    *sum += *kernel_elapsed;
}

// Check values of parameters in constant memory
__global__ void checkConstantValues(int* dev_equal,
    				    Grid* dev_grid,
				    Params* dev_params)
{

  // Values ok (0)
  *dev_equal = 0;

  // Compare values between global- and constant
  // memory structures
  if (dev_grid->nd != devC_grid.nd ||
      dev_grid->origo[0] != devC_grid.origo[0] ||
      dev_grid->origo[1] != devC_grid.origo[1] ||
      dev_grid->origo[2] != devC_grid.origo[2] ||
      dev_grid->L[0] != devC_grid.L[0] ||
      dev_grid->L[1] != devC_grid.L[1] ||
      dev_grid->L[2] != devC_grid.L[2] ||
      dev_grid->num[0] != devC_grid.num[0] ||
      dev_grid->num[1] != devC_grid.num[1] ||
      dev_grid->num[2] != devC_grid.num[2])
    *dev_equal = 1; // Not ok

  else if (dev_params->global != devC_params.global ||
      dev_params->g[0] != devC_params.g[0] ||
      dev_params->g[1] != devC_params.g[1] ||
      dev_params->g[2] != devC_params.g[2] ||
      dev_params->dt != devC_params.dt ||
      dev_params->np != devC_params.np ||
      dev_params->nw != devC_params.nw ||
      dev_params->wmode[0] != devC_params.wmode[0] ||
      dev_params->k_n != devC_params.k_n ||
      dev_params->k_t != devC_params.k_t ||
      dev_params->k_r != devC_params.k_r ||
      dev_params->gamma_n != devC_params.gamma_n ||
      dev_params->gamma_t != devC_params.gamma_t ||
      dev_params->gamma_r != devC_params.gamma_r ||
      dev_params->gamma_wn != devC_params.gamma_wn ||
      dev_params->gamma_wt != devC_params.gamma_wt ||
      dev_params->gamma_wr != devC_params.gamma_wr ||
      dev_params->mu_s != devC_params.mu_s ||
      dev_params->mu_d != devC_params.mu_d ||
      dev_params->mu_r != devC_params.mu_r ||
      dev_params->rho != devC_params.rho ||
      dev_params->kappa != devC_params.kappa ||
      dev_params->db != devC_params.db ||
      dev_params->V_b != devC_params.V_b ||
      dev_params->periodic != devC_params.periodic ||
      dev_params->shearmodel != devC_params.shearmodel)
    *dev_equal = 2; // Not ok

}


// Copy the constant data components to device memory,
// and check whether the values correspond to the 
// values in constant memory.
__host__ void DEM::checkConstantMemory()
{

  //cudaPrintfInit();

  // Allocate space in global device memory
  Grid* dev_grid;
  Params* dev_params;
  hipMalloc((void**)&dev_grid, sizeof(Grid));
  hipMalloc((void**)&dev_params, sizeof(Params));

  // Copy structure data from host to global device memory
  hipMemcpy(dev_grid, grid, sizeof(Grid), hipMemcpyHostToDevice);
  hipMemcpy(dev_params, params, sizeof(Params), hipMemcpyHostToDevice);

  // Compare values between global and constant memory
  // structures on the device.
  int* equal = new int;	// The values are equal = 0, if not = 1
  *equal = 0;
  int* dev_equal;
  hipMalloc((void**)&dev_equal, sizeof(int));
  checkConstantValues<<<1,1>>>(dev_equal, dev_grid, dev_params);
  checkForCudaErrors("After constant memory check");

  // Copy result to host
  hipMemcpy(equal, dev_equal, sizeof(int), hipMemcpyDeviceToHost);

  // Free global device memory
  hipFree(dev_grid);
  hipFree(dev_params);
  hipFree(dev_equal);

  //cudaPrintfDisplay(stdout, true);

  // Are the values equal?
  if (*equal != 0) {
    std::cerr << "Error! The values in constant memory do not "
              << "seem to be correct (" << *equal << ").\n";
    exit(1);
  } else {
    std::cout << "  Constant values ok (" << *equal << ").\n";
  }
}

// Copy selected constant components to constant device memory.
__host__ void DEM::transferToConstantDeviceMemory()
{
  using std::cout;

  if (verbose == 1)
    cout << "\n  Transfering data to constant device memory:     ";

  hipMemcpyToSymbol(HIP_SYMBOL("devC_nd"), &nd, sizeof(nd));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_np"), &np, sizeof(np));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_nc"), &NC, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("devC_dt"), &time.dt, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(devC_grid), grid, sizeof(Grid));
  hipMemcpyToSymbol(HIP_SYMBOL(devC_params), params, sizeof(Params));
  
  checkForCudaErrors("After transferring to device constant memory");
  
  if (verbose == 1)
    cout << "Done\n";

  //checkConstantMemory();
}


// Allocate device memory for particle variables,
// tied to previously declared pointers in structures
__host__ void DEM::allocateGlobalDeviceMemory(void)
{
  // Particle memory size
  unsigned int memSizeF  = sizeof(Float) * np;
  unsigned int memSizeF4 = sizeof(Float4) * np;

  if (verbose == 1)
    std::cout << "  Allocating device memory:                       ";

  // Particle arrays
  hipMalloc((void**)&dev_k.x, memSizeF4);
  hipMalloc((void**)&dev_sort.x_sorted, memSizeF4);
  hipMalloc((void**)&dev_k.vel, memSizeF4);
  hipMalloc((void**)&dev_sort.vel_sorted, memSizeF4);
  hipMalloc((void**)&dev_k.angvel, memSizeF4);
  hipMalloc((void**)&dev_sort.angvel_sorted, memSizeF4);
  hipMalloc((void**)&dev_k.acc, memSizeF4);
  k.acc = new Float4[np];
  hipMalloc((void**)&dev_k.angacc, memSizeF4);
  k.angacc = new Float4[np];
  hipMalloc((void**)&dev_k.force, memSizeF4);
  hipMalloc((void**)&dev_k.torque, memSizeF4);
  hipMalloc((void**)&dev_k.angpos, memSizeF4);
  hipMalloc((void**)&dev_e.es_dot, memSizeF);
  hipMalloc((void**)&dev_e.ev_dot, memSizeF);
  hipMalloc((void**)&dev_e.es, memSizeF);
  hipMalloc((void**)&dev_e.ev, memSizeF);
  hipMalloc((void**)&dev_e.p, memSizeF);

  // Cell-related arrays
  hipMalloc((void**)&dev_sort.gridParticleCellID, sizeof(unsigned int)*np);
  hipMalloc((void**)&dev_sort.gridParticleIndex, sizeof(unsigned int)*np);
  hipMalloc((void**)&dev_sort.cellStart, sizeof(unsigned int)*grid.num[0]*grid.num[1]*grid.num[2]);
  hipMalloc((void**)&dev_sort.cellEnd, sizeof(unsigned int)*grid.num[0]*grid.num[1]*grid.num[2]);

  // Particle contact bookkeeping arrays
  hipMalloc((void**)&dev_k.contacts, sizeof(unsigned int)*np*NC); // Max NC contacts per particle
  hipMalloc((void**)&dev_k.distmod, sizeof(Float4)*np*NC);
  hipMalloc((void**)&dev_k.delta_t, sizeof(Float4)*np*NC);

  // Host contact bookkeeping arrays
  k.contacts = new unsigned int[np*NC];
  // Initialize contacts lists to np
  for (unsigned int i=0; i<(np*NC); ++i)
    k.contacts[i] = np;
  k.distmod = new Float4[np*NC];
  k.delta_t = new Float4[np*NC];

  // Wall arrays
  hipMalloc((void**)&dev_walls.nx, sizeof(Float4)*walls.nw);
  hipMalloc((void**)&dev_walls.mvfd, sizeof(Float4)*walls.nw);
  hipMalloc((void**)&dev_walls.force, sizeof(Float)*walls.nw*np);
  // dev_w_force_partial allocated later

  checkForCudaErrors("End of allocateGlobalDeviceMemory");
  if (verbose == 1)
    std::cout << "Done\n";
}

__host__ void DEM::freeGlobalDeviceMemory()
{
  if (verbose == 1)
    printf("\nLiberating device memory:                        ");
  // Particle arrays
  hipFree(dev_k.x);
  hipFree(dev_sort.x_sorted);
  hipFree(dev_k.vel);
  hipFree(dev_sort.vel_sorted);
  hipFree(dev_k.angvel);
  hipFree(dev_sort.angvel_sorted);
  hipFree(dev_k.acc);
  hipFree(dev_k.angacc);
  hipFree(dev_k.force);
  hipFree(dev_k.torque);
  hipFree(dev_k.angpos);
  hipFree(dev_e.es_dot);
  hipFree(dev_e.ev_dot);
  hipFree(dev_e.es);
  hipFree(dev_e.ev);
  hipFree(dev_e.p);
  hipFree(dev_k.contacts);
  hipFree(dev_k.distmod);
  hipFree(dev_k.delta_t);

  // Cell-related arrays
  hipFree(dev_sort.gridParticleIndex);
  hipFree(dev_sort.cellStart);
  hipFree(dev_sort.cellEnd);

  // Wall arrays
  hipFree(dev_walls.nx);
  hipFree(dev_walls.mvfd);
  hipFree(dev_walls.force);
  hipFree(dev_w_force_partial);

  if (verbose == 1)
    printf("Done\n");
}


__host__ void DEM::transferToGlobalDeviceMemory()
{
  if (verbose == 1)
    std::cout << "  Transfering data to the device:                 ";

  // Copy structure data from host to global device memory
  /*hipMemcpy(dev_k, k, sizeof(Kinematics), hipMemcpyHostToDevice);
  hipMemcpy(dev_e, e, sizeof(Energies), hipMemcpyHostToDevice);
  hipMemcpy(dev_time, time, sizeof(Time), hipMemcpyHostToDevice);
  hipMemcpy(dev_walls, walls, sizeof(Walls), hipMemcpyHostToDevice);*/
  hipMemcpy(dev_k, k, sizeof(k), hipMemcpyHostToDevice);
  hipMemcpy(dev_e, e, sizeof(e), hipMemcpyHostToDevice);
  hipMemcpy(dev_time, time, sizeof(time), hipMemcpyHostToDevice);
  hipMemcpy(dev_walls, walls, sizeof(walls), hipMemcpyHostToDevice);

  checkForCudaErrors("End of transferToGlobalDeviceMemory");
  if (verbose == 1)
    std::cout << "Done\n";
}

__host__ void DEM::transferToGlobalDeviceMemory()
{
  cout << "  Transfering data to the device:                 ";

  // Copy structure data from host to global device memory
  hipMemcpy(k, dev_k, sizeof(k), hipMemcpyDeviceToHost);
  hipMemcpy(e, dev_e, sizeof(e), hipMemcpyDeviceToHost);
  hipMemcpy(time, dev_time, sizeof(time), hipMemcpyDeviceToHost);
  hipMemcpy(walls, dev_walls, sizeof(walls), hipMemcpyDeviceToHost);

  checkForCudaErrors("End of transferFromGlobalDeviceMemory");
  if (verbose == 1)
    std::cout << "Done\n";
}


// Iterate through time by explicit time integration
__host__ void DEM::startTime()
{

  using std::cout; // Namespace directive
  char file[200];  // Output filename
  FILE *fp;

  // Copy data to constant global device memory
  transferToConstantDeviceMemory();


  // Particle memory size
  unsigned int memSizeF  = sizeof(Float) * np;
  unsigned int memSizeF4 = sizeof(Float4) * np;

  // Allocate device memory for particle variables,
  // tied to previously declared pointers in structures
  allocateGlobalDeviceMemory();

  // Transfer data from host to gpu device memory
  transferToGlobalDeviceMemory();

  // Synchronization point
  hipDeviceSynchronize();
  checkForCudaErrors("Start of startTime()");

  // Model world variables
  float tic, toc, filetimeclock, time_spent, dev_time_spent;

  // Start CPU clock
  tic = clock();

  // GPU workload configuration
  unsigned int threadsPerBlock = 256; 
  // Create enough blocks to accomodate the particles
  unsigned int blocksPerGrid   = iDivUp(p.np, threadsPerBlock); 
  dim3 dimGrid(blocksPerGrid, 1, 1); // Blocks arranged in 1D grid
  dim3 dimBlock(threadsPerBlock, 1, 1); // Threads arranged in 1D block
  // Shared memory per block
  unsigned int smemSize = sizeof(unsigned int)*(threadsPerBlock+1);

  Float* dev_w_force_partial;
  hipMalloc((void**)&dev_w_force_partial, sizeof(Float)*dimGrid.x);

  // Report to stdout
  if (verbose == 1) {
    cout << "\n  Device memory allocation and transfer complete.\n"
      << "  - Blocks per grid: "
      << dimGrid.x << "*" << dimGrid.y << "*" << dimGrid.z << "\n"
      << "  - Threads per block: "
      << dimBlock.x << "*" << dimBlock.y << "*" << dimBlock.z << "\n"
      << "  - Shared memory required per block: " << smemSize << " bytes\n";
  }

  // Initialize counter variable values
  filetimeclock = 0.0;
  long iter = 0;

  // Create first status.dat
  sprintf(file,"output/%s.status.dat", inputbin);
  fp = fopen(file, "w");
  fprintf(fp,"%2.4e %2.4e %d\n", 
      	  time.current, 
	  100.0*time.current/time.total, 
	  time.step_count);
  fclose(fp);

  // Write first output data file: output0.bin, thus testing writing of bin files
  sprintf(file,"output/%s.output0.bin", inputbin);
  writebin(file);

  if (verbose == 1) {
    cout << "\n  Entering the main calculation time loop...\n\n"
      << "  IMPORTANT: Do not close this terminal, doing so will \n"
      << "             terminate this SPHERE process. Follow the \n"
      << "             progress by executing:\n"
      << "                $ ./sphere_status " << inputbin << "\n\n";
  }


  // Start GPU clock
  hipEvent_t dev_tic, dev_toc;
  hipEventCreate(&dev_tic);
  hipEventCreate(&dev_toc);
  hipEventRecord(dev_tic, 0);

  // If profiling is enabled, initialize timers for each kernel
  hipEvent_t kernel_tic, kernel_toc;
  float kernel_elapsed;
  double t_calcParticleCellID = 0.0;
  double t_thrustsort = 0.0;
  double t_reorderArrays = 0.0;
  double t_topology = 0.0;
  double t_interact = 0.0;
  double t_integrate = 0.0;
  double t_summation = 0.0;
  double t_integrateWalls = 0.0;

  if (PROFILING == 1) {
    hipEventCreate(&kernel_tic);
    hipEventCreate(&kernel_toc);
  }

  cout << "  Current simulation time: " << time.current << " s.";


  // MAIN CALCULATION TIME LOOP
  while (time.current <= time.total) {

    // Increment iteration counter
    ++iter;

    // Print current step number to terminal
    //printf("Step: %d\n", time.step_count);


    // Routine check for errors
    checkForCudaErrors("Start of main while loop");


    // For each particle: 
    // Compute hash key (cell index) from position 
    // in the fine, uniform and homogenous grid.
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    calcParticleCellID<<<dimGrid, dimBlock>>>(dev_sort.gridParticleCellID, 
					      dev_sort.gridParticleIndex, 
					      dev_k.x);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_calcParticleCellID);
    checkForCudaErrors("Post calcParticleCellID");


    // Sort particle (key, particle ID) pairs by hash key with Thrust radix sort
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    thrust::sort_by_key(thrust::device_ptr<uint>(dev_sort.gridParticleCellID),
			thrust::device_ptr<uint>(dev_sort.gridParticleCellID + np),
			thrust::device_ptr<uint>(dev_sort.gridParticleIndex));
    hipDeviceSynchronize(); // Needed? Does thrust synchronize threads implicitly?
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_thrustsort);
    checkForCudaErrors("Post thrust::sort_by_key");


    // Zero cell array values by setting cellStart to its highest possible value,
    // specified with pointer value 0xffffffff, which for a 32 bit unsigned int
    // is 4294967295.
    hipMemset(dev_sort.cellStart, 0xffffffff, 
	       grid.num[0]*grid.num[1]*grid.num[2]*sizeof(unsigned int));
    hipDeviceSynchronize();
    checkForCudaErrors("Post hipMemset");

    // Use sorted order to reorder particle arrays (position, velocities, radii) to ensure
    // coherent memory access. Save ordered configurations in new arrays (*_sorted).
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    reorderArrays<<<dimGrid, dimBlock, smemSize>>>(dev_sort.cellStart, 
						   dev_sort.cellEnd,
						   dev_sort.gridParticleCellID, 
						   dev_sort.gridParticleIndex,
						   dev_k.x, dev_k.vel, 
						   dev_k.angvel,
						   dev_sort.x_sorted, 
						   dev_sort.vel_sorted, 
						   dev_sort.angvel_sorted);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_reorderArrays);
    checkForCudaErrors("Post reorderArrays", iter);

    // The contact search in topology() is only necessary for determining
    // the accumulated shear distance needed in the linear elastic
    // and nonlinear contact force model
    if (params.contactmodel == 2 || params.contactmodel == 3) {
      // For each particle: Search contacts in neighbor cells
      if (PROFILING == 1)
	startTimer(&kernel_tic);
      topology<<<dimGrid, dimBlock>>>(dev_sort.cellStart, 
				      dev_sort.cellEnd,
				      dev_sort.gridParticleIndex,
				      dev_sort.x_sorted, 
				      dev_k.contacts,
				      dev_k.distmod);


      // Synchronization point
      hipDeviceSynchronize();
      if (PROFILING == 1)
	stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_topology);
      checkForCudaErrors("Post topology: One or more particles moved outside the grid.\nThis could possibly be caused by a numerical instability.\nIs the computational time step too large?", iter);
    }


    // For each particle: Process collisions and compute resulting forces.
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    interact<<<dimGrid, dimBlock>>>(dev_sort.gridParticleIndex,
				    dev_sort.cellStart,
				    dev_sort.cellEnd,
				    dev_k.x,
				    dev_sort.x_sorted,
				    dev_sort.vel_sorted,
				    dev_sort.angvel_sorted,
				    dev_k.vel,
				    dev_k.angvel,
				    dev_k.force, 
				    dev_k.torque,
				    dev_e.es_dot,
				    dev_e.ev_dot, 
				    dev_e.es, dev_e.ev, dev_e.p,
				    dev_walls.nx,
				    dev_walls.mvfd,
				    dev_walls.force,
				    dev_k.contacts,
				    dev_k.distmod,
				    dev_k.delta_t);


    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_interact);
    checkForCudaErrors("Post interact - often caused if particles move outside the grid", iter);

    // Update particle kinematics
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    integrate<<<dimGrid, dimBlock>>>(dev_sort.x_sorted, 
				     dev_sort.vel_sorted, 
				     dev_sort.angvel_sorted,
				     dev_k.x, 
				     dev_k.vel, 
				     dev_k.angvel,
				     dev_k.force,
				     dev_k.torque, 
				     dev_k.angpos,
				     dev_sort.gridParticleIndex);

    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_integrate);

    // Summation of forces on wall
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    summation<<<dimGrid, dimBlock>>>(dev_walls.force, dev_w_force_partial);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_summation);
    checkForCudaErrors("Post integrate & wall force summation");

    // Update wall kinematics
    if (PROFILING == 1)
      startTimer(&kernel_tic);
    integrateWalls<<< 1, walls.nw>>>(dev_walls.nx, 
				     dev_walls.mvfd,
				     dev_w_force_partial,
				     blocksPerGrid);

    // Synchronization point
    hipDeviceSynchronize();
    if (PROFILING == 1)
      stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed, &t_integrateWalls);
    checkForCudaErrors("Post integrateWalls");


    // Update timers and counters
    time.current  += time.dt;
    filetimeclock += time.dt;

    // Report time to console
    if (verbose == 1) {
      cout << "\r  Current simulation time: " 
	<< time.current << " s.        ";// << std::flush;
    }


    // Produce output binary if the time interval 
    // between output files has been reached
    if (filetimeclock > time.file_dt) {

      // Pause the CPU thread until all CUDA calls previously issued are completed
      hipDeviceSynchronize();
      checkForCudaErrors("Beginning of file output section");

      //// Copy device data to host memory
      transferFromGlobalDeviceMemory();

      // Pause the CPU thread until all CUDA calls previously issued are completed
      hipDeviceSynchronize();

      // Write binary output file
      time.step_count += 1;
      sprintf(file,"output/%s.output%d.bin", inputbin, time.step_count);
      writebin(file);


      if (CONTACTINFO == 1) {
	// Write contact information to stdout
	cout << "\n\n---------------------------\n"
	     << "t = " << time.current << " s.\n"
	     << "---------------------------\n";

	for (int n = 0; n < p.np; ++n) {
	  cout << "\n## Particle " << n << " ##\n";

	  cout  << "- contacts:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << "[" << nc << "]=" << host_contacts[nc+NC*n] << '\n';

	  cout << "\n- delta_t:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << host_delta_t[nc+NC*n].x << '\t'
		 << host_delta_t[nc+NC*n].y << '\t'
		 << host_delta_t[nc+NC*n].z << '\t'
		 << host_delta_t[nc+NC*n].w << '\n';

	  cout << "\n- distmod:\n";
	  for (int nc = 0; nc < NC; ++nc) 
	    cout << host_distmod[nc+NC*n].x << '\t'
		 << host_distmod[nc+NC*n].y << '\t'
		 << host_distmod[nc+NC*n].z << '\t'
		 << host_distmod[nc+NC*n].w << '\n';
	}
	cout << '\n';
      }

      // Update status.dat at the interval of filetime 
      sprintf(file,"output/%s.status.dat", inputbin);
      fp = fopen(file, "w");
      fprintf(fp,"%2.4e %2.4e %d\n", 
	      time.current, 
	      100.0*time.current/time.total,
	      time.step_count);
      fclose(fp);

      filetimeclock = 0.0;
    }
  }

  // Stop clock and display calculation time spent
  toc = clock();
  hipEventRecord(dev_toc, 0);
  hipEventSynchronize(dev_toc);

  time_spent = (toc - tic)/(CLOCKS_PER_SEC);
  hipEventElapsedTime(&dev_time_spent, dev_tic, dev_toc);

  cout << "\nSimulation ended. Statistics:\n"
       << "  - Last output file number: " 
       << time.step_count << "\n"
       << "  - GPU time spent: "
       << dev_time_spent/1000.0f << " s\n"
       << "  - CPU time spent: "
       << time_spent << " s\n"
       << "  - Mean duration of iteration:\n"
       << "      " << dev_time_spent/((double)iter*1000.0f) << " s\n"; 

  hipEventDestroy(dev_tic);
  hipEventDestroy(dev_toc);

  hipEventDestroy(kernel_tic);
  hipEventDestroy(kernel_toc);

  // Report time spent on each kernel
  if (PROFILING == 1) {
    double t_sum = t_calcParticleCellID + t_thrustsort + t_reorderArrays
                 + t_topology + t_interact + t_summation + t_integrateWalls;
    cout << "\nKernel profiling statistics:\n"
         << "  - calcParticleCellID:\t" << t_calcParticleCellID/1000.0 << " s"
	 << "\t(" << 100.0*t_calcParticleCellID/t_sum << " %)\n"
         << "  - thrustsort:\t\t" << t_thrustsort/1000.0 << " s"
	 << "\t(" << 100.0*t_thrustsort/t_sum << " %)\n"
         << "  - reorderArrays:\t" << t_reorderArrays/1000.0 << " s"
	 << "\t(" << 100.0*t_reorderArrays/t_sum << " %)\n"
         << "  - topology:\t\t" << t_topology/1000.0 << " s"
	 << "\t(" << 100.0*t_topology/t_sum << " %)\n"
         << "  - interact:\t\t" << t_interact/1000.0 << " s"
	 << "\t(" << 100.0*t_interact/t_sum << " %)\n"
         << "  - integrate:\t\t" << t_integrate/1000.0 << " s"
	 << "\t(" << 100.0*t_integrate/t_sum << " %)\n"
         << "  - summation:\t\t" << t_summation/1000.0 << " s"
	 << "\t(" << 100.0*t_summation/t_sum << " %)\n"
         << "  - integrateWalls:\t" << t_integrateWalls/1000.0 << " s"
	 << "\t(" << 100.0*t_integrateWalls/t_sum << " %)\n";
  }


  // Free GPU device memory  
  freeGlobalDeviceMemory();

  // Contact info arrays
  delete[] host_contacts;
  delete[] host_distmod;
  delete[] host_delta_t;

} /* EOF */
